
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 1024

int A_arr[N];
int B_arr[N];
int C_arr[N];

extern __device__ int
add1(int a, int b);

__global__ void add(const int *A, const int *B, int *C, size_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
	C[i] = add1(A[i], B[i]);
	printf("i = %d, A[%d] = %d, B[%d] = %d, C[%d] = %d, blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d\n", i, i, A[i], i,
	B[i], i, C[i], blockIdx.x, blockDim.x, threadIdx.x);
    }
}

int main(void) {
    for (size_t i = 0; i < N; i++)
	A_arr[i] = i, B_arr[i] = 1;

    int *dev_A, *dev_B, *dev_C;
    assert(hipSuccess == hipMalloc(&dev_A, N * sizeof(int)));
    assert(hipSuccess == hipMalloc(&dev_B, N * sizeof(int)));
    assert(hipSuccess == hipMalloc(&dev_C, N * sizeof(int)));
    assert(hipSuccess == hipMemcpy(dev_A, A_arr, N * sizeof(int), hipMemcpyHostToDevice));
    assert(hipSuccess == hipMemcpy(dev_B, B_arr, N * sizeof(int), hipMemcpyHostToDevice));

    size_t blockDim_x = 512;
    size_t nr_block = (N + blockDim_x - 1) / blockDim_x;

    add<<<nr_block, blockDim_x>>>(dev_A, dev_B, dev_C, N);
    assert(hipSuccess == hipMemcpy(C_arr, dev_C, N * sizeof(int), hipMemcpyDeviceToHost));

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    printf("C = {");
    for (size_t i = 0; i < N; i++)
	printf("%d,", C_arr[i]);
    printf("}\n");

    return 0;
}
